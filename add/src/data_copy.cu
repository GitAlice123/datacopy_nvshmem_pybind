#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <nvshmem.h>
#include <nvshmemx.h>
#include <stdio.h>
#include <assert.h>
#include <torch/extension.h>
#include <vector>
#include <cstring>
#include <atomic>
#include "ibgda_device.cuh"
#include "double_buffer_manager.h"
__global__ void send_recv_clean_kernel(DoubleBufferState state, int dst_pe);

__global__ void ibgda_initialize_recv_queue(int rank) {
    auto thread_idx = static_cast<int>(threadIdx.x);
    auto num_threads = static_cast<int>(blockDim.x);

    auto dst_rank = static_cast<int>(blockIdx.x);
    if (dst_rank != rank) {
        for (int qp_id = thread_idx; qp_id < datacopy::ibgda_get_state()->num_rc_per_pe; qp_id += num_threads) {
            auto qp = datacopy::ibgda_get_rc(dst_rank, qp_id);

            // Clean some necessary variables
            for (int i = 0; i < qp->rx_wq.nwqes; ++ i)
                datacopy::ibgda_write_empty_recv_wqe(datacopy::ibgda_get_wqe_ptr(qp, i));
            qp->mvars.rx_wq.resv_head = 0;
            qp->mvars.rx_wq.cons_idx = 0;

            // Allocate receive slots
            datacopy::nvshmemi_ibgda_allocate_recvs(qp);
        }
    }
}

int init_nvshmem(const std::vector<uint8_t> &root_unique_id_val, int rank, int world_size) {
    nvshmemx_uniqueid_t root_unique_id;
    nvshmemx_init_attr_t attr;
    
    std::memcpy(&root_unique_id, root_unique_id_val.data(), sizeof(nvshmemx_uniqueid_t));
    
    nvshmemx_set_attr_uniqueid_args(rank, world_size, &root_unique_id, &attr);
    nvshmemx_init_attr(NVSHMEMX_INIT_WITH_UNIQUEID, &attr);

    nvshmemi_device_host_state_t* dev_state_ptr = nullptr;
    CUDA_CHECK(hipGetSymbolAddress(reinterpret_cast<void**>(&dev_state_ptr), nvshmemi_device_state_d));

    bool ibgda_is_initialized = false;
    hipMemcpy(&dev_state_ptr->ibgda_is_initialized, &ibgda_is_initialized, sizeof(bool), hipMemcpyHostToDevice);    

    ibgda_initialize_recv_queue<<<world_size, 128>>>(rank);
    
    nvshmem_barrier_all();
    return nvshmem_my_pe();
}


std::vector<uint8_t> get_unique_id() {
    nvshmemx_uniqueid_t unique_id;
    nvshmemx_get_uniqueid(&unique_id);
    std::vector<uint8_t> result(sizeof(nvshmemx_uniqueid_t));
    std::memcpy(result.data(), &unique_id, sizeof(nvshmemx_uniqueid_t));
    return result;
}

// 实现 DoubleBufferManager 的方法
DoubleBufferManager::DoubleBufferManager() : initialized(false), n_tokens(0), token_size(0) {}

DoubleBufferManager::~DoubleBufferManager() {
    cleanup();
}

void DoubleBufferManager::init(int n_tokens, int token_size) {
    if (initialized) {
        cleanup();
    }
    this->n_tokens = n_tokens;
    this->token_size = token_size;
    init_double_buffer(&state, n_tokens, token_size);
    initialized = true;
}

void DoubleBufferManager::cleanup() {
    if (initialized) {
        cleanup_double_buffer(&state);
        initialized = false;
    }
}

void DoubleBufferManager::test_bandwidth(int rank, int world_size) {
    if (!initialized) {
        throw std::runtime_error("DoubleBufferManager not initialized");
    }
    int dst_pe = (rank == 0) ? 1 : 0;
    
    // 计算线程配置
    int total_need_threads = n_tokens * 2;
    int block_size = 256;
    int grid_size = (total_need_threads + block_size - 1) / block_size;
    
    // 启动内核
    send_recv_clean_kernel<<<grid_size, block_size>>>(state, dst_pe);
    
    // 等待内核完成
    hipDeviceSynchronize();

    // 切换缓冲区
    state.current_buffer = 1 - state.current_buffer;
}

void DoubleBufferManager::init_double_buffer(DoubleBufferState* state, int n_tokens, int token_size) {
    state->n_tokens = n_tokens;
    state->token_size = token_size;
    state->current_buffer = 0;
    
    // 检查参数有效性
    if (n_tokens <= 0 || token_size <= 0) {
        fprintf(stderr, "Error: Invalid parameters: n_tokens=%d, token_size=%d\n", n_tokens, token_size);
        exit(EXIT_FAILURE);
    }
    
    // 分配缓冲区0
    state->buffer0_send = (char*)nvshmem_malloc(n_tokens * token_size);
    state->buffer0_recv = (char*)nvshmem_malloc(n_tokens * token_size);
    state->buffer0_signals = (volatile int*)nvshmem_malloc(n_tokens * sizeof(int));
    
    // 检查分配是否成功
    if (!state->buffer0_send || !state->buffer0_recv || !state->buffer0_signals) {
        fprintf(stderr, "Error: Failed to allocate buffer0\n");
        exit(EXIT_FAILURE);
    }
    
    // 分配缓冲区1
    state->buffer1_send = (char*)nvshmem_malloc(n_tokens * token_size);
    state->buffer1_recv = (char*)nvshmem_malloc(n_tokens * token_size);
    state->buffer1_signals = (volatile int*)nvshmem_malloc(n_tokens * sizeof(int));
    
    if (!state->buffer1_send || !state->buffer1_recv || !state->buffer1_signals) {
        fprintf(stderr, "Error: Failed to allocate buffer1\n");
        exit(EXIT_FAILURE);
    }

    state->buffer_tmp_signals = (volatile int*)nvshmem_malloc(n_tokens * sizeof(int));
    
    if (!state->buffer_tmp_signals) {
        fprintf(stderr, "Error: Failed to allocate buffer_tmp_signals\n");
        exit(EXIT_FAILURE);
    }
    
    // 在主机上创建临时缓冲区并初始化
    int* host_signals = (int*)malloc(n_tokens * sizeof(int));
    if (!host_signals) {
        fprintf(stderr, "Error: Failed to allocate host_signals\n");
        exit(EXIT_FAILURE);
    }
    
    for (int i = 0; i < n_tokens; ++i) {
        host_signals[i] = 0;
    }
    
    // 使用 hipMemcpy 将数据从主机复制到设备
    hipError_t err;
    
    err = hipMemcpy((void*)state->buffer0_signals, host_signals, n_tokens * sizeof(int), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "hipMemcpy error (buffer0_signals): %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    err = hipMemcpy((void*)state->buffer1_signals, host_signals, n_tokens * sizeof(int), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "hipMemcpy error (buffer1_signals): %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    err = hipMemcpy((void*)state->buffer_tmp_signals, host_signals, n_tokens * sizeof(int), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "hipMemcpy error (buffer_tmp_signals): %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    // 释放主机内存
    free(host_signals);
    
    // 确保内存同步
    nvshmem_barrier_all();
}

void DoubleBufferManager::cleanup_double_buffer(DoubleBufferState* state) {
    nvshmem_free(state->buffer0_send);
    nvshmem_free(state->buffer0_recv);
    nvshmem_free((void*)state->buffer0_signals);
    
    nvshmem_free(state->buffer1_send);
    nvshmem_free(state->buffer1_recv);
    nvshmem_free((void*)state->buffer1_signals);

    nvshmem_free((void*)state->buffer_tmp_signals);
}

// 内核函数实现
__global__ void send_recv_clean_kernel(DoubleBufferState state, int dst_pe) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int total_threads = gridDim.x * blockDim.x;
    
    // 确定当前和下一个缓冲区
    int current_idx = state.current_buffer;
    int next_idx = 1 - current_idx;
    
    // 获取当前缓冲区指针
    char* send_buffer = (current_idx == 0) ? state.buffer0_send : state.buffer1_send;
    char* recv_buffer = (current_idx == 0) ? state.buffer0_recv : state.buffer1_recv;
    volatile int* signals = (current_idx == 0) ? state.buffer0_signals : state.buffer1_signals;
    
    // 获取下一个缓冲区指针（用于清理）
    volatile int* next_signals = (next_idx == 0) ? state.buffer0_signals : state.buffer1_signals;
    
    // 线程分组：前一半用于通信，后一半用于清理
    int comm_threads = total_threads / 2;
    int clean_threads = comm_threads;
    
    if (tid < comm_threads) {
        // 通信线程：执行发送和接收
        if (tid < state.n_tokens) {
            // 发送数据
            char* token_data_send = send_buffer + tid * state.token_size;
            char* token_data_recv = recv_buffer + tid * state.token_size;
            
            datacopy::nvshmemi_ibgda_put_nbi_thread(
                (uint64_t)token_data_recv, (uint64_t)token_data_send, state.token_size, dst_pe, tid, tid);
            
            // 发送信号
            int signal = tid + 1;  // 信号值 = token 索引 + 1
            datacopy::nvshmemi_ibgda_amo_nonfetch_add((void*)&signals[tid], signal, dst_pe, tid, false);
            
            // 轮询接收信号 - 使用直接访问而不是 __ldg
            while (signals[tid] != tid + 1);
        }
    } else {
        // 清理线程：清理下一个缓冲区
        int clean_tid = tid - comm_threads;
        for(int i = clean_tid; i < state.n_tokens; i += clean_threads) {
            next_signals[i] = 0; // 重置信号
        }
    }
}